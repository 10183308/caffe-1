#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>

#include <stdio.h>
#include <stdlib.h>

#include <sys/types.h>
#include <dirent.h>
#include <boost/lexical_cast.hpp>
#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>

#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/DenseBlock_layer.hpp"

namespace caffe {

  bool dirExists_cu(string dirStr){
    const char* dirCStr = dirStr.c_str();
    DIR* dir = opendir(dirCStr);
    if (ENOENT == errno){
      return false;
    }
    closedir(dir);
    return true;
  }

  void tryCreateDirectory_cu(string fileName){
    vector<string> strVec;
    boost::split(strVec,fileName,boost::is_any_of("/"));
    string newStr="";
    for (int i=0;i<strVec.size()-1;++i){
      newStr += strVec[i] + (i==strVec.size()-2?"":"/");
    }
    boost::filesystem::path dirToCreate(newStr);
    if (!dirExists_cu(newStr)){
      boost::filesystem::create_directories(dirToCreate);
    }
  }


string itos_cu(int i){
  string output = boost::lexical_cast<string>(i);
  return output; 
}

template <typename Dtype>
void gpu_copy_one_to_many(const Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_input,int chunkStride_output){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
	const Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkSize_input; 
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkStride_output;
       
	//printf("inpointer %p\n",inPtr_gpu);
	//printf("outpointer %p\n",outPtr_gpu);
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_input * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void gpu_copy_many_to_one(Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_output,int chunkStride_input){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
        Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkStride_input;
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkSize_output;
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_output * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void print_gpuPtr(Dtype* gpuPtr,int numValues){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    for (int i=0;i<numValues;++i){
      std::cout<< cpuPtr[i] <<",";
    }
    std::cout<<std::endl;
}

template <typename Dtype>
void log_gpuPtr(Dtype* gpuPtr,int numValues,string fileName){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    const char* fileName_cstr = fileName.c_str();
    tryCreateDirectory_cu(fileName_cstr);
    std::ofstream outWriter(fileName_cstr,std::ofstream::out);
    for (int i=0;i<numValues;++i){
      outWriter<<cpuPtr[i]<<",";
    }
    outWriter<<std::endl;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::logInternal_gpu(string dir){
    string localDir = dir+"/gpu_"+itos_cu(this->logId)+"/";
    int postBufferSize = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
    //postConv_data_gpu
    log_gpuPtr(this->postConv_data_gpu,postBufferSize,localDir+"postConv_data_gpu");
    //postConv_grad_gpu
    log_gpuPtr(this->postConv_grad_gpu,postBufferSize,localDir+"postConv_grad_gpu");
    //postBN_data_gpu
    log_gpuPtr(this->postBN_data_gpu,postBufferSize,localDir+"postBN_data_gpu");
    //postBN_grad_gpu
    log_gpuPtr(this->postBN_grad_gpu,postBufferSize,localDir+"postBN_grad_gpu");
    //postReLU_data_gpu
    log_gpuPtr(this->postReLU_data_gpu,postBufferSize,localDir+"postReLU_data_gpu");
    //postReLU_grad_gpu
    log_gpuPtr(this->postReLU_grad_gpu,postBufferSize,localDir+"postReLU_grad_gpu");
    //ResultRunningMean_gpu
    int numChannelsTotal = this->initChannel + this->growthRate * this->numTransition;
    log_gpuPtr(this->ResultRunningMean_gpu,numChannelsTotal,localDir+"ResultRunningMean_gpu");
    //ResultRunningVariance_gpu
    log_gpuPtr(this->ResultRunningVariance_gpu,numChannelsTotal,localDir+"ResultRunningVariance_gpu");
    for (int transitionIdx=0;transitionIdx<this->numTransition;++transitionIdx){
      //Filter_grad_gpu
      int filterSize = (this->initChannel+this->growthRate*transitionIdx) * this->growthRate * this->filter_H * this->filter_W;
      log_gpuPtr(this->blobs_[transitionIdx]->mutable_gpu_diff(),filterSize,localDir+"Filter_grad_gpu_"+itos_cu(transitionIdx));
      //Scaler_grad_gpu
      int numChannelLocal = transitionIdx==0?this->initChannel:this->growthRate;
      log_gpuPtr(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_diff(),numChannelLocal,localDir+"Scaler_grad_gpu_"+itos_cu(transitionIdx));
      log_gpuPtr(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_data(),numChannelLocal,localDir+"Scaler_data_gpu_"+itos_cu(transitionIdx));
      //Bias_grad_gpu
      log_gpuPtr(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_diff(),numChannelLocal,localDir+"Bias_grad_gpu_"+itos_cu(transitionIdx));
      log_gpuPtr(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_data(),numChannelLocal,localDir+"Bias_data_gpu_"+itos_cu(transitionIdx));
    }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::GPU_Initialization(){
    //GPU intermediate ptrs
    int bufferSize_byte = this->N*(this->initChannel+this->growthRate*this->numTransition)*this->H*this->W*sizeof(Dtype);
    CUDA_CHECK(hipMalloc(&this->postConv_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postConv_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_grad_gpu,bufferSize_byte));

    hipMemset(this->postConv_data_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_data_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_data_gpu,0,bufferSize_byte);
    hipMemset(this->postConv_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_grad_gpu,0,bufferSize_byte);
    //workspace
    CUDA_CHECK(hipMalloc(&this->workspace,this->workspace_size_bytes));
    hipMemset(this->workspace,0,this->workspace_size_bytes);
        
    //handles and descriptors
    //cudnn handle
    this->cudnnHandlePtr = new hipdnnHandle_t;
    CUDNN_CHECK(hipdnnCreate(this->cudnnHandlePtr));
    //conv_y global tensor descriptor
    this->tensorDescriptor_conv_y = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y,this->N,this->growthRate,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);	
    //per transition variables
    for (int i=0;i<this->numTransition;++i){
	//Result Running/Saving Mean/Variance/InvVariance
    	int localChannel = this->initChannel + i * this->growthRate;
    	Dtype* local_ResultMean;
	Dtype* local_ResultVar;
	Dtype* local_SaveMean;
	Dtype* local_SaveInvVar;
	
	CUDA_CHECK(hipMalloc(&local_ResultMean,localChannel*sizeof(Dtype)));
    	CUDA_CHECK(hipMalloc(&local_ResultVar,localChannel*sizeof(Dtype)));
    	CUDA_CHECK(hipMalloc(&local_SaveMean,localChannel*sizeof(Dtype)));
    	CUDA_CHECK(hipMalloc(&local_SaveInvVar,localChannel*sizeof(Dtype)));
		
    	hipMemset(local_ResultMean,0,localChannel*sizeof(Dtype));
    	hipMemset(local_ResultVar,0,localChannel*sizeof(Dtype));
    	hipMemset(local_SaveMean,0,localChannel*sizeof(Dtype));
    	hipMemset(local_SaveInvVar,0,totalChannel*sizeof(Dtype));
   
	this->ResultRunningMean_gpu.push_back(local_ResultMean);
	this->ResultRunningVariance_gpu.push_back(local_ResultVar);
	this->ResultSaveMean_gpu.push_back(local_SaveMean);
	this->ResultSaveInvVariance_gpu.push_back(local_SaveInvVar);
	//Result Save Variance[i] for i-th transition
	int reversibleChannels=(i==0)?0:initChannel+(i-1)*growthRate;
	Dtype* local_SaveVar;
	CUDA_CHECK(hipMalloc(&local_SaveVar,reversibleChannels*sizeof(Dtype)));
	hipMemset(local_SaveVar,0,reversibleChannels*sizeof(Dtype));
	this->ResultSaveVariance_gpu.push_back(local_SaveVar);
	
	//narrow descriptor
	int narrowChannelNum = (i==0?this->initChannel:this->growthRate);
	hipdnnTensorDescriptor_t * narrow_Desc_local = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(narrow_Desc_local);
	cudnn::setTensor4dDesc<Dtype>(narrow_Desc_local,this->N,narrowChannelNum,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);
	this->tensorDescriptorVec_narrow.push_back(narrow_Desc_local);
	//conv_x descriptor
	int conv_x_channels = this->initChannel + this->growthRate * i;
	hipdnnTensorDescriptor_t * wide_Desc_local_x = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(wide_Desc_local_x);
	cudnn::setTensor4dDesc<Dtype>(wide_Desc_local_x,this->N,conv_x_channels,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);
	this->tensorDescriptorVec_conv_x.push_back(wide_Desc_local_x); 
	//filter Descriptor for Convolution
	hipdnnFilterDescriptor_t * localFilterDesc = new hipdnnFilterDescriptor_t;
	cudnn::createFilterDesc<Dtype>(localFilterDesc,growthRate,conv_x_channels,this->filter_H,this->filter_W);
	this->filterDescriptorVec.push_back(localFilterDesc);
	//wide channelwise descriptor for BN type2
	int channelsBefore_noself = i==0?0:initChannel + (i-1) * growthRate;
	hipdnnTensorDescriptor_t * wide_BNparam = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(wide_BNparam);
	cudnn::setTensor4dDesc<Dtype>(1,channelsBefore_noself,1,1);
	this->tensorDescriptor_BN_wide.push_back(wide_BNparam);
    }
    //BN parameter (Scale,Bias) Descriptor
    this->tensorDescriptor_BN_initChannel = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_BN_initChannel);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_BN_initChannel,1,this->initChannel,1,1);
    this->tensorDescriptor_BN_growthRate = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_BN_growthRate);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_BN_growthRate,1,this->growthRate,1,1);
    //Conv Descriptor
    this->conv_Descriptor = new hipdnnConvolutionDescriptor_t;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(this->conv_Descriptor));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(*this->conv_Descriptor,this->pad_h,this->pad_w,this->conv_verticalStride,this->conv_horizentalStride,1,1,HIPDNN_CONVOLUTION));

}

template <typename Dtype>
void cleanupBuffer(Dtype* ptr_gpu,int count){
    hipMemset(ptr_gpu,0,count*sizeof(Dtype));
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::LoopEndCleanup_gpu(){
    int valsBuffer = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
    cleanupBuffer(this->postConv_data_gpu,valsBuffer);
    cleanupBuffer(this->postConv_grad_gpu,valsBuffer);
    cleanupBuffer(this->postBN_data_gpu,valsBuffer);
    cleanupBuffer(this->postBN_grad_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_data_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_grad_gpu,valsBuffer);
}

template <typename Dtype>
__global__ void helper_computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W,int channelLimit){
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx =  (index / (H * W)) % (initChannel + growthRate * numTransition);
    if (localChannelIdx < channelLimit){
      caffe_gpu_atomic_add((xPtr[index]-batchMeanPtr[localChannelIdx]) * (xPtr[index]-batchMeanPtr[localChannelIdx]),batchVarPtr + localChannelIdx);
    }
  }
}

//variance is only used in the reverse BN process
template <typename Dtype>
void computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){ 
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate; 
  helper_computeBatchVariance<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>((n,xPtr,batchMeanPtr,batchVarPtr,transitionIdx,numTransition,N,initChannel,growthRate,H,W,channelLimit);
  int M = N * H * W;
  caffe_gpu_scal<Dtype>(channelLimit,1.0/(M-1),batchVarPtr);
}

//ReLU: Negative_slope = 0.5
template <typename Dtype>
__global__ void ReLUForward(int n,Dtype* xPtr,Dtype* yPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index / (H * W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, fwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      yPtr[index] = xPtr[index] > 0? xPtr[index]: 0.5 * xPtr[index]; 
    }
  }
}

template <typename Dtype>
__global__ void ReLUBackward(int n,Dtype* xPtr,Dtype* dxPtr,Dtype* dyPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, bwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      dxPtr[index] = xPtr[index]>0?dyPtr[index]:0.5*dyPtr[index];
    }
  }
}

template <typename Dtype>
__global__ void ReLUReverse(int n,Dtype* yPtr,Dtype* xPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition); 
    //i.e. for transitionIdx==1, only reverse transform region 0
    if (localChannelIdx < channelLimit){
      xPtr[index] = yPtr[index]>=0?yPtr[index]:2*yPtr[index];
    }
  }
}

template <typename Dtype>
__global__ void BNReverse(int n,Dtype* yPtr,Dtype* xPtr,Dtype* scalerPtr,Dtype* biasPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,double epsilon,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition); 
    //i.e. for transitionIdx==1, only reverse transform region 0
    if (localChannelIdx < channelLimit){
      //x = a * y + b :: affine transform, find out a and b
      double a = sqrt(batchVarPtr[localChannelIdx] + epsilon) / (scalerPtr[localChannelIdx]);
      double b = batchMeanPtr[localChannelIdx] - (biasPtr[localChannelIdx]*sqrt(batchVarPtr[localChannelIdx]+epsilon))/(scalerPtr[localChannelIdx]);
      xPtr[index] = a * yPtr[index] + b;
    }
  }
}

//acts on cpu pointer
template <typename Dtype>
void ScalerProtector(Dtype* scaler_mutable_data,int numValues){
  for (int i=0;i<numValues;++i){
    Dtype localVal = scaler_mutable_data[i];
    if ((localVal < 1e-3) && (localVal > -1e-3)){
        if (localVal>=0){scaler_mutable_data[i] = 1e-3;}
	else {scaler_mutable_data[i] = -1e-3;}
    }
  }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (!this->gpuInited){
      //std::cout<< "start gpu init"<<std::endl;
      this->GPU_Initialization();
      this->gpuInited = true;
      //std::cout<< "end gpu init"<<std::endl;
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  //copy to bottom_data to buffer with stride
  int chunkSize_copy_init = this->initChannel * this->H * this->W;
  int chunkStride_copy = (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
  gpu_copy_one_to_many<Dtype>(bottom_data,this->postConv_data_gpu,this->N,chunkSize_copy_init,chunkStride_copy);
  //work in the buffer, transition by transition
  for (int transitionIdx=0;transitionIdx < this->numTransition;++transitionIdx){
      //use scaler protector before forward
      int totalLocalChannel = initChannel + growthRate * transitionIdx;
      ScalerProtector(this->blobs_[numTransition + transitionIdx]->mutable_cpu_data(),totalLocalChannel);
      //BN::type1 normal narrow channels::postConv -> postBN 
      int channelsBefore_noself = (transitionIdx==0?0:(this->initChannel + (transitionIdx - 1)*this->growthRate));
      Dtype* BN_narrow_x_ptr = this->postConv_data_gpu + channelsBefore_noself * this->H * this->W;  
      Dtype* BN_narrow_y_ptr = this->postBN_data_gpu + channelsBefore_noself * this->H * this->W;
      Dtype* BN_narrow_globalMean= this->ResultRunningMean_gpu[transitionIdx] + channelsBefore_noself;
      Dtype* BN_narrow_globalVar = this->ResultRunningVariance_gpu[transitionIdx] + channelsBefore_noself;
      hipdnnTensorDescriptor_t * narrowBN_paramDesc = (transitionIdx==0?tensorDescriptor_BN_initChannel:tensorDescriptor_BN_growthRate);
      
      if (this->phase_ == TEST){
          CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_y_ptr,
	    *narrowBN_paramDesc,
	    this->blobs_[this->numTransition+transitionIdx]->gpu_data()+channelsBefore_noself,
            this->blobs_[2*this->numTransition+transitionIdx]->gpu_data()+channelsBefore_noself,
	    BN_narrow_globalMean,BN_narrow_globalVar,HIPDNN_BN_MIN_EPSILON)
	  );
      }
      else{
          Dtype* batchMean = this->ResultSaveMean_gpu[transitionIdx] + channelsBefore_noself;
          Dtype* batchInvVar =  this->ResultSaveInvVariance_gpu[transitionIdx] + channelsBefore_noself;
	  double EMA_factor = 1.0/(1+this->trainCycleIdx);
	  CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_y_ptr,
	    *narrowBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data() + channelsBefore_noself,
	    this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data() + channelsBefore_noself,
	    EMA_factor,BN_narrow_globalMean,BN_narrow_globalVar,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	  );
      }
      //BN :: type2: wide channels, for anything prior to channels for
      //type1 BN
      if (transitionIdx > 0){
        hipdnnTensorDescriptor_t wideBN_paramDesc = this->tensorDescriptor_BN_wide[transitionIdx]; 
	Dtype* BN_wide_x_ptr = this->postReLU_data_gpu;
	Dtype* BN_wide_y_ptr = this->postBN_data_gpu;
	Dtype* BN_wide_globalMean = this->ResultRunningMean_gpu;
	Dtype* BN_wide_globalVar = this->ResultRunningVariance_gpu;
	
	if (this->phase_ == TEST){
          CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_y_ptr,
	    *wideBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
            this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    BN_wide_globalMean,BN_wide_globalVar,HIPDNN_BN_MIN_EPSILON)
	  );
	}
	else {
          Dtype* batchMean = this->ResultSaveMean_gpu[transitionIdx];
	  Dtype* batchInvVar = this->ResultSaveInvVariance_gpu[transitionIdx];
          double EMA_factor = 1.0/(1+this->trainCycleIdx);
          CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_y_ptr,
	    *wideBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	    this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    EMA_factor,BN_wide_globalMean,BN_wide_globalVar,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	  );
	}
        //compute per channel variance after BN
        int work_n = this->N * (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W; 
        computeBatchVariance<Dtype><<<CAFFE_GET_BLOCKS(work_n), CAFFE_CUDA_NUM_THREADS>>>(work_n,this->postReLU_data_gpu,this->ResultSaveMean_gpu[transitionIdx],this->ResultSaveVariance_gpu[transitionIdx],transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
      }

      //ReLU
      Dtype* ReLU_x_ptr = this->postBN_data_gpu;
      Dtype* ReLU_y_ptr = this->postReLU_data_gpu;
      ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(work_n), CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_ptr,ReLU_y_ptr,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
      //Convolution
      int delayChannel = this->initChannel + this->growthRate * transitionIdx;
      Dtype* conv_x_local = this->postReLU_data_gpu;
      Dtype* conv_y_local = this->postConv_data_gpu + delayChannel * this->H * this->W;
      CUDNN_CHECK(hipdnnConvolutionForward(*(this->cudnnHandlePtr),
	cudnn::dataType<Dtype>::one,
	*(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	*(this->filterDescriptorVec[transitionIdx]),
	this->blobs_[transitionIdx]->gpu_data(),
	*(this->conv_Descriptor),HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
	this->workspace,this->workspace_size_bytes,cudnn::dataType<Dtype>::zero,
	*(this->tensorDescriptor_conv_y),conv_y_local	
	)		      
      ); 
  } 
  this->trainCycleIdx += 1;
  //change top data
  int numValues = this->N * (this->initChannel+this->growthRate*this->numTransition) * this->H * this->W; 
  CUDA_CHECK(hipMemcpy(top[0]->mutable_gpu_data(),this->postConv_data_gpu,numValues * sizeof(Dtype),hipMemcpyDeviceToDevice));
  //this->logInternal_gpu("TClog");
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    if (!this->gpuInited){
	this->GPU_Initialization();
    	this->gpuInited = true;
    }

    //assuming buffers store already computed value, always propagate down
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    //deploy top diff to buffer
    int numValues = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
    CUDA_CHECK(hipMemcpy(this->postConv_grad_gpu,top_diff,numValues * sizeof(Dtype),hipMemcpyDeviceToDevice));
    //Backward, transition by transition
    for (int transitionIdx=this->numTransition-1;transitionIdx>=0;--transitionIdx){
        int channelsBefore_self = this->initChannel + transitionIdx * this->growthRate;
        int channelsBefore_noself = transitionIdx>0?(this->initChannel + (transitionIdx - 1) * this->growthRate):0;
	//Conv
        Dtype* filterGrad_local = this->blobs_[transitionIdx]->mutable_gpu_diff();
	const Dtype* filterData_local =this->blobs_[transitionIdx]->gpu_data();
	Dtype* conv_x_local = postReLU_data_gpu;
	Dtype* conv_dy_local = postConv_grad_gpu + channelsBefore_self * this->H * this->W;
	//Conv w.r.t. filter
	CUDNN_CHECK(hipdnnConvolutionBackwardFilter(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one, 
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::zero,
	  *(this->filterDescriptorVec[transitionIdx]),filterGrad_local	  
	  )		
	);
	//Conv w.r.t. x
	CUDNN_CHECK(hipdnnConvolutionBackwardData(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one,
	  *(this->filterDescriptorVec[transitionIdx]),filterData_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::one,
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),postReLU_grad_gpu
	  )		
	);
	//ReLU Bwd, for any j <= transitionIdx
	int work_n = this->N * (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;
	Dtype* ReLU_x_local = postBN_data_gpu;
	Dtype* ReLU_dy_local = postReLU_grad_gpu;
        Dtype* ReLU_dx_local = postBN_grad_gpu;	
	ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_local,ReLU_dx_local,ReLU_dy_local,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
        //ReLU Reverse, for any j < transitionIdx
        Dtype* ReLU_reverse_y_local = postBN_data_gpu;
	Dtype* ReLU_reverse_x_local = postReLU_data_gpu;
	ReLUReverse<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_reverse_y_local,ReLU_reverse_x_local,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
	//BN Bwd, type2, wide
	if (transitionIdx > 0){
	  Dtype* BNwide_x_local = this->postReLU_data_gpu;
	  Dtype* BNwide_dx_local = this->postReLU_grad_gpu;
	  Dtype* BNwide_dy_local = this->postBN_grad_gpu;
	  Dtype* saveMeanwide_local = this->ResultSaveMean_gpu[transitionIdx]; 
	  Dtype* saveInvVarwide_local = this->ResultSaveInvVariance_gpu[transitionIdx];
	  hipdnnTensorDescriptor_t * BNwideparam_desc = this->tensorDescriptor_BN_wide[transitionIdx];
	
	  CUDNN_CHECK(hipdnnBatchNormalizationBackward(*(this->cudnnHandlePtr),
	    HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_x_local,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_dy_local,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_dx_local,
	    *BNwideparam_desc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	    this->blobs_[this->numTransition + transitionIdx]->mutable_gpu_diff(),
	    this->blobs_[2*this->numTransition + transitionIdx]->mutable_gpu_diff(),
	    HIPDNN_BN_MIN_EPSILON,saveMeanwide_local,saveInvVarwide_local
	    )		
	  );
	}
	//BN Bwd, type1, narrow
        Dtype* BNnarrow_x_local = this->postConv_data_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* BNnarrow_dx_local = this->postConv_grad_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* BNnarrow_dy_local = this->postBN_grad_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* saveMeannarrow_local = this->ResultSaveMean_gpu + channelsBefore_noself;
	Dtype* saveInvVarnarrow_local = this->ResultSaveInvVariance_gpu + channelsBefore_noself;
        hipdnnTensorDescriptor_t * BNnarrowparam_desc = (transitionIdx==0)?tensorDescriptor_BN_initChannel : tensorDescriptor_BN_growthRate;
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(*(this->cudnnHandlePtr),
	  HIPDNN_BATCHNORM_SPATIAL,
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_x_local,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_dy_local,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_dx_local,
	  *BNnarrowparam_desc,
	  this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	  this->blobs_[this->numTransition + transitionIdx]->mutable_gpu_diff(),
	  this->blobs_[2*this->numTransition + transitionIdx]->mutable_gpu_diff(),
	  HIPDNN_BN_MIN_EPSILON,saveMeannarrow_local,saveInvVarnarrow_local
	  )		
	);	
	//BN reverse
        Dtype* BN_reverse_y_local = this->postReLU_data_gpu;
	Dtype* BN_reverse_x_local = this->postBN_data_gpu;
	Dtype* scalerPtr = this->blobs_[this->numTransition+transitionIdx]->gpu_data() + channelsBefore_noself;
	Dtype* biasPtr = this->blobs_[2*this->numTransition+transitionIdx]->gpu_data() + channelsBefore_noself;
	Dtype* batchMeanPtr = this->ResultSaveMean_gpu[transitionIdx] + channelsBefore_noself;
	Dtpye* batchVarPtr = this->ResultSaveVariance_gpu[transitionIdx] + channelsBefore_noself;
	BNReverse<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,BN_reverse_y_local,BN_reverse_x_local,scalerPtr,biasPtr,batchMeanPtr,batchVarPtr,HIPDNN_BN_MIN_EPSILON,this->N,this->initChannel,this->growthRate,this->H,this->W);
    }
    //deploy buffer to bottom diff 
    int chunkSize_copy_init = this->initChannel * this->H * this->W;
    int chunkStride_copy = (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;
    gpu_copy_many_to_one(postConv_grad_gpu,bottom_diff,this->N,chunkSize_copy_init,chunkStride_copy);
    //this->logInternal_gpu("TClog");
    this->LoopEndCleanup_gpu();
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

}  // namespace caffe
