#include "hip/hip_runtime.h"
#include <time.h>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>

#include <stdio.h>
#include <stdlib.h>

#include <sys/types.h>
#include <dirent.h>
#include <boost/lexical_cast.hpp>
#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>

#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/DenseBlock_layer.hpp"

namespace caffe {

  bool dirExists_cu(string dirStr){
    const char* dirCStr = dirStr.c_str();
    DIR* dir = opendir(dirCStr);
    if (ENOENT == errno){
      return false;
    }
    closedir(dir);
    return true;
  }

  void tryCreateDirectory_cu(string fileName){
    vector<string> strVec;
    boost::split(strVec,fileName,boost::is_any_of("/"));
    string newStr="";
    for (int i=0;i<strVec.size()-1;++i){
      newStr += strVec[i] + (i==strVec.size()-2?"":"/");
    }
    boost::filesystem::path dirToCreate(newStr);
    if (!dirExists_cu(newStr)){
      boost::filesystem::create_directories(dirToCreate);
    }
  }


string itos_cu(int i){
  string output = boost::lexical_cast<string>(i);
  return output; 
}

template <typename Dtype>
void gpu_copy_one_to_many(const Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_input,int chunkStride_output){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
	const Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkSize_input; 
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkStride_output;
       
	//printf("inpointer %p\n",inPtr_gpu);
	//printf("outpointer %p\n",outPtr_gpu);
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_input * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void gpu_copy_many_to_one(Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_output,int chunkStride_input){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
        Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkStride_input;
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkSize_output;
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_output * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void print_gpuPtr(Dtype* gpuPtr,int numValues){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    for (int i=0;i<numValues;++i){
      std::cout<< cpuPtr[i] <<",";
    }
    std::cout<<std::endl;
}

template <typename Dtype>
void log_gpuPtr(Dtype* gpuPtr,int numValues,string fileName){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    const char* fileName_cstr = fileName.c_str();
    tryCreateDirectory_cu(fileName_cstr);
    std::ofstream outWriter(fileName_cstr,std::ofstream::out);
    for (int i=0;i<numValues;++i){
      outWriter<<cpuPtr[i]<<",";
    }
    outWriter<<std::endl;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::logInternal_gpu(string dir,int TIdx,bool logDynamic,bool logDiff){
    string localDir = dir+"/gpu_"+itos_cu(this->logId)+"/";
    if (logDynamic){
      int postBufferSize = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
      if (logDiff){
        //postConv_grad_gpu
        log_gpuPtr<Dtype>(this->postConv_grad_gpu,postBufferSize,localDir+"postConv_grad_gpu_transition"+itos_cu(TIdx));
        //postBN_grad_gpu
        log_gpuPtr<Dtype>(this->postBN_grad_gpu,postBufferSize,localDir+"postBN_grad_gpu_transition"+itos_cu(TIdx));
        //postReLU_grad_gpu
        log_gpuPtr<Dtype>(this->postReLU_grad_gpu,postBufferSize,localDir+"postReLU_grad_gpu_transition"+itos_cu(TIdx));
      }
      else {
        //postConv_data_gpu
        log_gpuPtr<Dtype>(this->postConv_data_gpu,postBufferSize,localDir+"postConv_data_gpu_transition"+itos_cu(TIdx));
        //postBN_data_gpu
        log_gpuPtr<Dtype>(this->postBN_data_gpu,postBufferSize,localDir+"postBN_data_gpu_transition"+itos_cu(TIdx));
        //postReLU_data_gpu
        log_gpuPtr<Dtype>(this->postReLU_data_gpu,postBufferSize,localDir+"postReLU_data_gpu_transition"+itos_cu(TIdx));
      }
    }
    else {
      for (int transitionIdx=0;transitionIdx<this->numTransition;++transitionIdx){
	int numChannel_moreWide = this->initChannel + this->growthRate * transitionIdx;
        //global/batch Mean/Variance
        log_gpuPtr<Dtype>(this->blobs_[3*this->numTransition+transitionIdx]->mutable_gpu_data(),numChannel_moreWide,localDir+"globalMean_gpu_transition"+itos_cu(transitionIdx));
        log_gpuPtr<Dtype>(this->blobs_[4*this->numTransition+transitionIdx]->mutable_gpu_data(),numChannel_moreWide,localDir+"globalVariance_gpu_transition"+itos_cu(transitionIdx));
      	log_gpuPtr<Dtype>(this->ResultSaveMean_gpu[transitionIdx],numChannel_moreWide,localDir+"ResultSaveMean_gpu_transition"+itos_cu(transitionIdx));
        log_gpuPtr<Dtype>(this->ResultSaveInvVariance_gpu[transitionIdx],numChannel_moreWide,localDir+"ResultSaveInvVariance_gpu_transition"+itos_cu(transitionIdx));
        //Filter_data/grad_gpu
        int filterSize = (this->initChannel+this->growthRate*transitionIdx) * this->growthRate * this->filter_H * this->filter_W;
        log_gpuPtr<Dtype>(this->blobs_[transitionIdx]->mutable_gpu_data(),filterSize,localDir+"Filter_data_gpu_"+itos_cu(transitionIdx));
	log_gpuPtr<Dtype>(this->blobs_[transitionIdx]->mutable_gpu_diff(),filterSize,localDir+"Filter_grad_gpu_"+itos_cu(transitionIdx));
        //Scaler_grad_gpu
        log_gpuPtr<Dtype>(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_diff(),numChannel_moreWide,localDir+"Scaler_grad_gpu_"+itos_cu(transitionIdx));
        log_gpuPtr<Dtype>(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_data(),numChannel_moreWide,localDir+"Scaler_data_gpu_"+itos_cu(transitionIdx));
        //Bias_grad_gpu
        log_gpuPtr<Dtype>(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_diff(),numChannel_moreWide,localDir+"Bias_grad_gpu_"+itos_cu(transitionIdx));
        log_gpuPtr<Dtype>(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_data(),numChannel_moreWide,localDir+"Bias_data_gpu_"+itos_cu(transitionIdx));
      }
    }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::GPU_Initialization(){
    //GPU intermediate ptrs
    int bufferSize_byte = this->N*(this->initChannel+this->growthRate*this->numTransition)*this->H*this->W*sizeof(Dtype);
    CUDA_CHECK(hipMalloc(&this->postConv_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postConv_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_grad_gpu,bufferSize_byte));

    hipMemset(this->postConv_data_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_data_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_data_gpu,0,bufferSize_byte);
    hipMemset(this->postConv_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_grad_gpu,0,bufferSize_byte);
    //workspace
    CUDA_CHECK(hipMalloc(&this->workspace,this->workspace_size_bytes));
    hipMemset(this->workspace,0,this->workspace_size_bytes);
    //handles and descriptors
    //cudnn handle
    this->cudnnHandlePtr = new hipdnnHandle_t;
    CUDNN_CHECK(hipdnnCreate(this->cudnnHandlePtr));
    //conv_y global tensor descriptor
    this->tensorDescriptor_conv_y = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y,this->N,this->growthRate,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);	
    //per transition variables
    for (int i=0;i<this->numTransition;++i){
        int cache_size = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
        Dtype* localCache_cpu = new Dtype[cache_size];
        postReLU_cache_cpu.push_back(localCache_cpu);
	//Result Running/Saving Mean/Variance/InvVariance
    	int localChannel = this->initChannel + i * this->growthRate;
    	Dtype* local_SaveMean;
	Dtype* local_SaveInvVar;
	
	CUDA_CHECK(hipMalloc(&local_SaveMean,localChannel*sizeof(Dtype)));
    	CUDA_CHECK(hipMalloc(&local_SaveInvVar,localChannel*sizeof(Dtype)));
		
    	hipMemset(local_SaveMean,0,localChannel*sizeof(Dtype));
    	hipMemset(local_SaveInvVar,0,localChannel*sizeof(Dtype));
   
	this->ResultSaveMean_gpu.push_back(local_SaveMean);
	this->ResultSaveInvVariance_gpu.push_back(local_SaveInvVar);
	
	//conv_x descriptor
	int conv_x_channels = this->initChannel + this->growthRate * i;
	hipdnnTensorDescriptor_t * wide_Desc_local_x = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(wide_Desc_local_x);
	cudnn::setTensor4dDesc<Dtype>(wide_Desc_local_x,this->N,conv_x_channels,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);
	this->tensorDescriptorVec_conv_x.push_back(wide_Desc_local_x); 
	//filter Descriptor for Convolution
	hipdnnFilterDescriptor_t * localFilterDesc = new hipdnnFilterDescriptor_t;
	cudnn::createFilterDesc<Dtype>(localFilterDesc,growthRate,conv_x_channels,this->filter_H,this->filter_W);
	this->filterDescriptorVec.push_back(localFilterDesc);
	//BN channel-wise Descriptor
	int channelsBefore_self = initChannel+growthRate*i;
	hipdnnTensorDescriptor_t * BNparam = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(BNparam);
	cudnn::setTensor4dDesc<Dtype>(BNparam,1,channelsBefore_self,1,1);
	this->tensorDescriptor_BN.push_back(BNparam);
    }
    //Conv Descriptor
    this->conv_Descriptor = new hipdnnConvolutionDescriptor_t;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(this->conv_Descriptor));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(*this->conv_Descriptor,this->pad_h,this->pad_w,this->conv_verticalStride,this->conv_horizentalStride,1,1,HIPDNN_CONVOLUTION));

    //Mean and Var tmp
    int totalNumChannel = this->initChannel + this->growthRate * this->numTransition;
    CUDA_CHECK(hipMalloc(&this->Mean_tmp, totalNumChannel*sizeof(Dtype)));
    CUDA_CHECK(hipMalloc(&this->Var_tmp, totalNumChannel*sizeof(Dtype)));
}

template <typename Dtype>
void cleanupBuffer(Dtype* ptr_gpu,int count){
    hipMemset(ptr_gpu,0,count*sizeof(Dtype));
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::LoopEndCleanup_gpu(){
    int valsBuffer = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
    cleanupBuffer(this->postConv_data_gpu,valsBuffer);
    cleanupBuffer(this->postConv_grad_gpu,valsBuffer);
    cleanupBuffer(this->postBN_data_gpu,valsBuffer);
    cleanupBuffer(this->postBN_grad_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_data_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_grad_gpu,valsBuffer);
}
/*
template <typename Dtype>
__global__ void helper_computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W,int channelLimit){
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx =  (index / (H * W)) % (initChannel + growthRate * numTransition);
    if (localChannelIdx < channelLimit){
      caffe_gpu_atomic_add((xPtr[index]-batchMeanPtr[localChannelIdx]) * (xPtr[index]-batchMeanPtr[localChannelIdx]),batchVarPtr + localChannelIdx);
    }
  }
}

//variance is only used in the reverse BN process
template <typename Dtype>
void computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){ 
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate; 
  helper_computeBatchVariance<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n,xPtr,batchMeanPtr,batchVarPtr,transitionIdx,numTransition,N,initChannel,growthRate,H,W,channelLimit);
  int M = N * H * W;
  caffe_gpu_scal<Dtype>(channelLimit,1.0/(M-1),batchVarPtr);
}
*/

//ReLU: Negative_slope = 0.5
template <typename Dtype>
__global__ void ReLUForward(int n,Dtype* xPtr,Dtype* yPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index / (H * W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, fwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      yPtr[index] = xPtr[index] > 0? xPtr[index]: 0.5 * xPtr[index]; 
    }
  }
}

template <typename Dtype>
__global__ void ReLUBackward(int n,Dtype* xPtr,Dtype* dxPtr,Dtype* dyPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, bwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      dxPtr[index] = xPtr[index]>0?dyPtr[index]:0.5*dyPtr[index];
    }
  }
}

/*
template <typename Dtype>
__global__ void ReLUReverse(int n,Dtype* yPtr,Dtype* xPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition); 
    //i.e. for transitionIdx==1, only reverse transform region 0
    if (localChannelIdx < channelLimit){
      xPtr[index] = yPtr[index]>=0?yPtr[index]:2*yPtr[index];
    }
  }
}

template <typename Dtype>
__global__ void BNForwardInf(int n,Dtype* xPtr,Dtype* yPtr,Dtype* scalerPtr,Dtype* biasPtr,Dtype* globalMeanPtr,Dtype* globalVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate*numTransition);
    if (localChannelIdx < channelLimit){
      yPtr[index] = (scalerPtr[localChannelIdx] * ((xPtr[index]-globalMeanPtr[localChannelIdx])/sqrt(globalVarPtr[localChannelIdx] + 1e-5))) + biasPtr[localChannelIdx];
    }
  }
}

template <typename Dtype>
void composeFwdOutput(Dtype* output,Dtype* frontB,Dtype* backB,int N,int channelFront,int channelBack,int H,int W){
  for (int n=0;n<N;++n){
    int numValuesFront = channelFront*H*W;
    int numValuesBack = channelBack*H*W;
    int offsetFront = n * (channelFront + channelBack) * H * W;
    int offsetBack = offsetFront + numValuesFront;
    hipMemcpy(output+offsetFront,frontB+offsetFront,numValuesFront*sizeof(Dtype),hipMemcpyDeviceToDevice);
    hipMemcpy(output+offsetBack,backB+offsetBack,numValuesBack*sizeof(Dtype),hipMemcpyDeviceToDevice);
  }
}

template <typename Dtype>
void distributeBwdInput(Dtype* input,Dtype* frontB,Dtype* backB,int N,int channelFront,int channelBack,int H,int W){
  for (int n=0;n<N;++n){
    int numValuesFront = channelFront*H*W;
    int numValuesBack = channelBack*H*W;
    int offsetFront = n * (channelFront + channelBack) * H * W;
    int offsetBack = offsetFront + numValuesFront;
    hipMemcpy(frontB+offsetFront,input+offsetFront,numValuesFront*sizeof(Dtype),hipMemcpyDeviceToDevice);
    hipMemcpy(backB+offsetBack,input+offsetBack,numValuesBack*sizeof(Dtype),hipMemcpyDeviceToDevice);
  }
}
*/

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (!this->gpuInited){
      //std::cout<<"Initializing GPU local"<<std::endl;
      this->GPU_Initialization();
      this->gpuInited = true;
  }
  clock_t begin_fwd = std::clock();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  //copy to bottom_data to buffer with stride
  int chunkSize_copy_init = this->initChannel * this->H * this->W;
  int chunkStride_copy = (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
  gpu_copy_one_to_many<Dtype>(bottom_data,this->postConv_data_gpu,this->N,chunkSize_copy_init,chunkStride_copy);
  int work_n = this->N * (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;     
  //work in the buffer, transition by transition
  for (int transitionIdx=0;transitionIdx < this->numTransition;++transitionIdx){
      //BN Fwd 
      Dtype* BN_x_ptr = this->postConv_data_gpu;  
      Dtype* BN_y_ptr = this->postBN_data_gpu;
      Dtype* BN_globalMean= this->blobs_[3*this->numTransition+transitionIdx]->mutable_gpu_data();
      Dtype* BN_globalVar = this->blobs_[4*this->numTransition+transitionIdx]->mutable_gpu_data();
      hipdnnTensorDescriptor_t * BN_paramDesc = tensorDescriptor_BN[transitionIdx];
      int numChannels = initChannel + growthRate*transitionIdx;
      Dtype* local_MeanInf = this->Mean_tmp;
      Dtype* local_VarInf = this->Var_tmp;
	      
      if (this->phase_ == TEST){
          Dtype scale_factor = this->blobs_[5*this->numTransition]->cpu_data()[0] == 0 ? 0 : 1.0 / this->blobs_[5*this->numTransition]->cpu_data()[0];
	  caffe_gpu_scale(numChannels,scale_factor,BN_globalMean,local_MeanInf);
          caffe_gpu_scale(numChannels,scale_factor,BN_globalVar,local_VarInf);

	  CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_y_ptr,
	    *BN_paramDesc,
	    this->blobs_[this->numTransition+transitionIdx]->gpu_data(),
            this->blobs_[2*this->numTransition+transitionIdx]->gpu_data(),
	    local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON)
	  );
      }
      else{
          Dtype* batchMean = this->ResultSaveMean_gpu[transitionIdx];
          Dtype* batchInvVar =  this->ResultSaveInvVariance_gpu[transitionIdx];
	  CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_y_ptr,
	    *BN_paramDesc,
	    this->blobs_[this->numTransition+transitionIdx]->mutable_gpu_data(),
	    this->blobs_[2*this->numTransition+transitionIdx]->mutable_gpu_data(),
	    Dtype(1),local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	  );
	  //update global Mean/Var manually
          //Mean:
	  caffe_gpu_axpby(numChannels,Dtype(1),local_MeanInf,this->EMA_decay,BN_globalMean);
          //Var:
	  caffe_gpu_axpby(numChannels,Dtype(1),local_VarInf,this->EMA_decay,BN_globalVar);

      }
      cleanupBuffer(this->Mean_tmp,numChannels);
      cleanupBuffer(this->Var_tmp,numChannels);
      //ReLU
      Dtype* ReLU_x_ptr = this->postBN_data_gpu;
      Dtype* ReLU_y_ptr = this->postReLU_data_gpu;
      ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(work_n), CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_ptr,ReLU_y_ptr,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
      //Convolution
      int delayChannel = this->initChannel + this->growthRate * transitionIdx;
      Dtype* conv_x_local = this->postReLU_data_gpu;
      Dtype* conv_y_local = this->postConv_data_gpu + delayChannel * this->H * this->W;
      CUDNN_CHECK(hipdnnConvolutionForward(*(this->cudnnHandlePtr),
	cudnn::dataType<Dtype>::one,
	*(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	*(this->filterDescriptorVec[transitionIdx]),
	this->blobs_[transitionIdx]->gpu_data(),
	*(this->conv_Descriptor),HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
	this->workspace,this->workspace_size_bytes,cudnn::dataType<Dtype>::zero,
	*(this->tensorDescriptor_conv_y),conv_y_local	
	)		      
      );
      //this->logInternal_gpu("TClog",transitionIdx,true,false);
  } 
  if (this->phase_ == TRAIN){
    this->blobs_[5*this->numTransition]->mutable_cpu_data()[0] *= this->EMA_decay;
    this->blobs_[5*this->numTransition]->mutable_cpu_data()[0] += 1;
    this->trainCycleIdx+=1;
  }
  //deploy top data
  hipMemcpy(top[0]->mutable_gpu_data(),postConv_data_gpu,work_n*sizeof(Dtype),hipMemcpyDeviceToDevice); 
  //clock_t end_fwd = std::clock();
  //double elapsed_fwd = double(end_fwd - begin_fwd) / CLOCKS_PER_SEC;
  //std::cout<<"elapsed fwd gpu:"<<elapsed_fwd<<std::endl;
  //this->logInternal_gpu("TClog",-1,false,false);
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    if (!this->gpuInited){
	this->GPU_Initialization();
    	this->gpuInited = true;
    } 
    //clock_t begin_bwd = std::clock();
    //assuming buffers store already computed value, always propagate down
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    int work_n = N * (initChannel+growthRate*numTransition) * H * W;
    //deploy top diff
    hipMemcpy(postConv_grad_gpu,top[0]->mutable_gpu_diff(),work_n*sizeof(Dtype),hipMemcpyDeviceToDevice);
    //Backward, transition by transition
    for (int transitionIdx=this->numTransition-1;transitionIdx>=0;--transitionIdx){
        int channelsBefore_self = this->initChannel + transitionIdx * this->growthRate;
        //Using BN & ReLU Fwd to generate corresponding postBN,postReLU data for this transition 
        //BN Fwd
        Dtype* BN_x_ptr = postConv_data_gpu;
        Dtype* BN_y_ptr = postBN_data_gpu; 
        Dtype* BN_globalMean = this->blobs_[3*this->numTransition+transitionIdx]->mutable_gpu_data();
        Dtype* BN_globalVar = this->blobs_[4*this->numTransition+transitionIdx]->mutable_gpu_data();  
        hipdnnTensorDescriptor_t* BN_paramDesc = tensorDescriptor_BN[transitionIdx];
        int numChannels = initChannel+growthRate*transitionIdx;
        Dtype* local_MeanInf = Mean_tmp;
	Dtype* local_VarInf = Var_tmp;
        Dtype* batchMean = ResultSaveMean_gpu[transitionIdx];
        Dtype* batchInvVar = ResultSaveInvVariance_gpu[transitionIdx];
        
        CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_y_ptr,
	    *BN_paramDesc,
	    this->blobs_[this->numTransition+transitionIdx]->gpu_data(),
	    this->blobs_[2*this->numTransition+transitionIdx]->gpu_data(),
	    Dtype(1),local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	);
        cleanupBuffer(this->Mean_tmp,numChannels);
        cleanupBuffer(this->Var_tmp,numChannels);
        //ReLU Fwd
        Dtype* ReLU_x_ptr = postBN_data_gpu;
        Dtype* ReLU_y_ptr = postReLU_data_gpu;
        ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(work_n), CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_ptr,ReLU_y_ptr,transitionIdx,numTransition,N,initChannel,growthRate,H,W);  
	//Now do Bwd
        //Conv
        Dtype* filterGrad_local = this->blobs_[transitionIdx]->mutable_gpu_diff();
	const Dtype* filterData_local =this->blobs_[transitionIdx]->gpu_data();
	Dtype* conv_x_local = postReLU_data_gpu;
	Dtype* conv_dy_local = postConv_grad_gpu + channelsBefore_self * this->H * this->W;
	//Conv w.r.t. filter
	CUDNN_CHECK(hipdnnConvolutionBackwardFilter(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one, 
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::one,
	  *(this->filterDescriptorVec[transitionIdx]),filterGrad_local	  
	  )		
	);
	//Conv w.r.t. x
	CUDNN_CHECK(hipdnnConvolutionBackwardData(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one,
	  *(this->filterDescriptorVec[transitionIdx]),filterData_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::zero,
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),postReLU_grad_gpu
	  )		
	);
	//ReLU Bwd
	Dtype* ReLU_x_local = postBN_data_gpu;
	Dtype* ReLU_dy_local = postReLU_grad_gpu;
        Dtype* ReLU_dx_local = postBN_grad_gpu;	
	ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_local,ReLU_dx_local,ReLU_dy_local,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
	//BN Bwd
        Dtype* BN_x_local = this->postConv_data_gpu;
	Dtype* BN_dx_local = this->postConv_grad_gpu;
	Dtype* BN_dy_local = this->postBN_grad_gpu;
	Dtype* saveMean_local = this->ResultSaveMean_gpu[transitionIdx];
	Dtype* saveInvVar_local = this->ResultSaveInvVariance_gpu[transitionIdx];
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(*(this->cudnnHandlePtr),
	  HIPDNN_BATCHNORM_SPATIAL,
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::one,
#if CUDNN_VERSION >= 4005
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::one,
#endif	  
          *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_x_local,
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_dy_local,
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),BN_dx_local,
	  *BN_paramDesc,
	  this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	  this->blobs_[this->numTransition + transitionIdx]->mutable_gpu_diff(),
	  this->blobs_[2*this->numTransition + transitionIdx]->mutable_gpu_diff(),
	  HIPDNN_BN_MIN_EPSILON,saveMean_local,saveInvVar_local
	  )		
	);
        //this->logInternal_gpu("TClog",transitionIdx,true,false);
        //this->logInternal_gpu("TClog",transitionIdx,true,true);
    }
    //deploy buffer to bottom diff
    //this->logInternal_gpu("TClog",-1,false,false);
    int chunkSize_copy_init = this->initChannel * this->H * this->W;
    int chunkStride_copy = (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;
    gpu_copy_many_to_one(postConv_grad_gpu,bottom_diff,this->N,chunkSize_copy_init,chunkStride_copy);
    this->LoopEndCleanup_gpu();
    //clock_t end_bwd = std::clock();
    //double elapsed_bwd = double(end_bwd - begin_bwd) / CLOCKS_PER_SEC;
    //std::cout<<"elapsed bwd time:"<<elapsed_bwd<<std::endl;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu_public(const vector<Blob<Dtype>*>& bottom,const vector<Blob<Dtype>*>& top){
  this->Forward_gpu(bottom,top);
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu_public(const vector<Blob<Dtype>*>& top,const vector<bool>& propagate_down,const vector<Blob<Dtype>*>& bottom){
  this->Backward_gpu(top,propagate_down,bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

}  // namespace caffe
