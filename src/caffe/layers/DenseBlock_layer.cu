#include "hip/hip_runtime.h"
#include <vector>
#include "hipDNN.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

#include "caffe/layers/DenseBlock_layer.hpp"

namespace caffe {

__global__ void DenseBlockForward(const int n, const float* in, float* out) {
  CUDA_KERNEL_LOOP(index, n){
    out[index] = sin(in[index]);
  } 
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
    
  const float* bottom_data_me;
  hipMalloc(&bottom_data_me,count * sizeof(float)); 
  // NOLINT_NEXT_LINE(whitespace/operators)
  DenseBlockForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_FL_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void DenseBlockBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype sinx = out_data[index];
    out_diff[index] = in_diff[index] * cos(sinx);
  }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    DenseBlockBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

}  // namespace caffe
