#include "hip/hip_runtime.h"
#include <time.h>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>

#include <stdio.h>
#include <stdlib.h>

#include <sys/types.h>
#include <dirent.h>
#include <boost/lexical_cast.hpp>
#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>

#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/DenseBlock_layer.hpp"

namespace caffe {

  bool dirExists_cu(string dirStr){
    const char* dirCStr = dirStr.c_str();
    DIR* dir = opendir(dirCStr);
    if (ENOENT == errno){
      return false;
    }
    closedir(dir);
    return true;
  }

  void tryCreateDirectory_cu(string fileName){
    vector<string> strVec;
    boost::split(strVec,fileName,boost::is_any_of("/"));
    string newStr="";
    for (int i=0;i<strVec.size()-1;++i){
      newStr += strVec[i] + (i==strVec.size()-2?"":"/");
    }
    boost::filesystem::path dirToCreate(newStr);
    if (!dirExists_cu(newStr)){
      boost::filesystem::create_directories(dirToCreate);
    }
  }


string itos_cu(int i){
  string output = boost::lexical_cast<string>(i);
  return output; 
}

template <typename Dtype>
void gpu_copy_one_to_many(const Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_input,int chunkStride_output){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
	const Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkSize_input; 
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkStride_output;
       
	//printf("inpointer %p\n",inPtr_gpu);
	//printf("outpointer %p\n",outPtr_gpu);
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_input * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void gpu_copy_many_to_one(Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_output,int chunkStride_input){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
        Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkStride_input;
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkSize_output;
	CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_output * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void print_gpuPtr(Dtype* gpuPtr,int numValues){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    for (int i=0;i<numValues;++i){
      std::cout<< cpuPtr[i] <<",";
    }
    std::cout<<std::endl;
}

template <typename Dtype>
void log_gpuPtr(Dtype* gpuPtr,int numValues,string fileName){
    Dtype* cpuPtr = new Dtype[numValues];
    hipMemcpy(cpuPtr,gpuPtr,numValues*sizeof(Dtype),hipMemcpyDeviceToHost);
    const char* fileName_cstr = fileName.c_str();
    tryCreateDirectory_cu(fileName_cstr);
    std::ofstream outWriter(fileName_cstr,std::ofstream::out);
    for (int i=0;i<numValues;++i){
      outWriter<<cpuPtr[i]<<",";
    }
    outWriter<<std::endl;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::logInternal_gpu(string dir,int TIdx,bool logDynamic,bool logDiff){
    string localDir = dir+"/gpu_"+itos_cu(this->logId)+"/";
    if (logDynamic){
      int postBufferSize = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
      if (logDiff){
        //postConv_grad_gpu
        log_gpuPtr(this->postConv_grad_gpu,postBufferSize,localDir+"postConv_grad_gpu_transition"+itos_cu(TIdx));
        //postBN_grad_gpu
        log_gpuPtr(this->postBN_grad_gpu,postBufferSize,localDir+"postBN_grad_gpu_transition"+itos_cu(TIdx));
        //postReLU_grad_gpu
        log_gpuPtr(this->postReLU_grad_gpu,postBufferSize,localDir+"postReLU_grad_gpu_transition"+itos_cu(TIdx));
      }
      else {
        //postConv_data_gpu
        log_gpuPtr(this->postConv_data_gpu,postBufferSize,localDir+"postConv_data_gpu_transition"+itos_cu(TIdx));
        //postBN_data_gpu
        log_gpuPtr(this->postBN_data_gpu,postBufferSize,localDir+"postBN_data_gpu_transition"+itos_cu(TIdx));
        //postReLU_data_gpu
        log_gpuPtr(this->postReLU_data_gpu,postBufferSize,localDir+"postReLU_data_gpu_transition"+itos_cu(TIdx));
      }
    }
    else {
      for (int transitionIdx=0;transitionIdx<this->numTransition;++transitionIdx){
	int numChannel_moreWide = this->initChannel + this->growthRate * transitionIdx;
        //global/batch Mean/Variance
        log_gpuPtr(this->blobs_[3*this->numTransition+transitionIdx]->gpu_data(),numChannel_moreWide,localDir+"globalMean_gpu_transition"+itos_cu(transitionIdx));
        log_gpuPtr(this->blobs_[4*this->numTransition+transitionIdx]->gpu_data(),numChannel_moreWide,localDir+"globalVariance_gpu_transition"+itos_cu(transitionIdx));
      	log_gpuPtr(this->ResultSaveMean_gpu[transitionIdx],numChannel_moreWide,localDir+"ResultSaveMean_gpu_transition"+itos_cu(transitionIdx));
        log_gpuPtr(this->ResultSaveInvVariance_gpu[transitionIdx],numChannel_moreWide,localDir+"ResultSaveInvVariance_gpu_transition"+itos_cu(transitionIdx));
        //Filter_grad_gpu
        int filterSize = (this->initChannel+this->growthRate*transitionIdx) * this->growthRate * this->filter_H * this->filter_W;
        log_gpuPtr(this->blobs_[transitionIdx]->mutable_gpu_diff(),filterSize,localDir+"Filter_grad_gpu_"+itos_cu(transitionIdx));
        //Scaler_grad_gpu
        log_gpuPtr(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_diff(),numChannel_moreWide,localDir+"Scaler_grad_gpu_"+itos_cu(transitionIdx));
        log_gpuPtr(this->blobs_[transitionIdx+this->numTransition]->mutable_gpu_data(),numChannel_moreWide,localDir+"Scaler_data_gpu_"+itos_cu(transitionIdx));
        //Bias_grad_gpu
        log_gpuPtr(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_diff(),numChannel_moreWide,localDir+"Bias_grad_gpu_"+itos_cu(transitionIdx));
        log_gpuPtr(this->blobs_[transitionIdx+2*this->numTransition]->mutable_gpu_data(),numChannel_moreWide,localDir+"Bias_data_gpu_"+itos_cu(transitionIdx));
      }
    }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::GPU_Initialization(){
    //GPU intermediate ptrs
    int bufferSize_byte = this->N*(this->initChannel+this->growthRate*this->numTransition)*this->H*this->W*sizeof(Dtype);
    CUDA_CHECK(hipMalloc(&this->postConv_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_data_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postConv_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postBN_grad_gpu,bufferSize_byte));
    CUDA_CHECK(hipMalloc(&this->postReLU_grad_gpu,bufferSize_byte));

    hipMemset(this->postConv_data_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_data_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_data_gpu,0,bufferSize_byte);
    hipMemset(this->postConv_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postBN_grad_gpu,0,bufferSize_byte);
    hipMemset(this->postReLU_grad_gpu,0,bufferSize_byte);
    //workspace
    CUDA_CHECK(hipMalloc(&this->workspace,this->workspace_size_bytes));
    hipMemset(this->workspace,0,this->workspace_size_bytes);
    //handles and descriptors
    //cudnn handle
    this->cudnnHandlePtr = new hipdnnHandle_t;
    CUDNN_CHECK(hipdnnCreate(this->cudnnHandlePtr));
    //conv_y global tensor descriptor
    this->tensorDescriptor_conv_y = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_conv_y,this->N,this->growthRate,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);	
    //per transition variables
    for (int i=0;i<this->numTransition;++i){
        int cache_size = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
        Dtype* localCache_cpu = new Dtype[cache_size];
        postReLU_cache_cpu.push_back(localCache_cpu);
	//Result Running/Saving Mean/Variance/InvVariance
    	int localChannel = this->initChannel + i * this->growthRate;
    	Dtype* local_SaveMean;
	Dtype* local_SaveInvVar;
	
	CUDA_CHECK(hipMalloc(&local_SaveMean,localChannel*sizeof(Dtype)));
    	CUDA_CHECK(hipMalloc(&local_SaveInvVar,localChannel*sizeof(Dtype)));
		
    	hipMemset(local_SaveMean,0,localChannel*sizeof(Dtype));
    	hipMemset(local_SaveInvVar,0,localChannel*sizeof(Dtype));
   
	this->ResultSaveMean_gpu.push_back(local_SaveMean);
	this->ResultSaveInvVariance_gpu.push_back(local_SaveInvVar);
	
	//narrow descriptor
	int narrowChannelNum = (i==0?this->initChannel:this->growthRate);
	hipdnnTensorDescriptor_t * narrow_Desc_local = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(narrow_Desc_local);
	cudnn::setTensor4dDesc<Dtype>(narrow_Desc_local,this->N,narrowChannelNum,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);
	this->tensorDescriptorVec_narrow.push_back(narrow_Desc_local);
	//conv_x descriptor
	int conv_x_channels = this->initChannel + this->growthRate * i;
	hipdnnTensorDescriptor_t * wide_Desc_local_x = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(wide_Desc_local_x);
	cudnn::setTensor4dDesc<Dtype>(wide_Desc_local_x,this->N,conv_x_channels,this->H,this->W,(this->numTransition*this->growthRate+this->initChannel)*this->H*this->W,this->H*this->W,this->W,1);
	this->tensorDescriptorVec_conv_x.push_back(wide_Desc_local_x); 
	//filter Descriptor for Convolution
	hipdnnFilterDescriptor_t * localFilterDesc = new hipdnnFilterDescriptor_t;
	cudnn::createFilterDesc<Dtype>(localFilterDesc,growthRate,conv_x_channels,this->filter_H,this->filter_W);
	this->filterDescriptorVec.push_back(localFilterDesc);
	//wide channelwise descriptor for BN type2
	int channelsBefore_noself = i==0?0:initChannel + (i-1) * growthRate;
	hipdnnTensorDescriptor_t * wide_BNparam = new hipdnnTensorDescriptor_t;
	cudnn::createTensor4dDesc<Dtype>(wide_BNparam);
	if (i>0) cudnn::setTensor4dDesc<Dtype>(wide_BNparam,1,channelsBefore_noself,1,1);
	this->tensorDescriptor_BN_wide.push_back(wide_BNparam);
    }
    //BN parameter (Scale,Bias) Descriptor
    this->tensorDescriptor_BN_initChannel = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_BN_initChannel);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_BN_initChannel,1,this->initChannel,1,1);
    this->tensorDescriptor_BN_growthRate = new hipdnnTensorDescriptor_t;
    cudnn::createTensor4dDesc<Dtype>(this->tensorDescriptor_BN_growthRate);
    cudnn::setTensor4dDesc<Dtype>(this->tensorDescriptor_BN_growthRate,1,this->growthRate,1,1);
    //Conv Descriptor
    this->conv_Descriptor = new hipdnnConvolutionDescriptor_t;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(this->conv_Descriptor));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(*this->conv_Descriptor,this->pad_h,this->pad_w,this->conv_verticalStride,this->conv_horizentalStride,1,1,HIPDNN_CONVOLUTION));

    //Mean and Var tmp
    int totalNumChannel = this->initChannel + this->growthRate * this->numTransition;
    CUDA_CHECK(hipMalloc(&this->Mean_tmp, totalNumChannel*sizeof(Dtype)));
    CUDA_CHECK(hipMalloc(&this->Var_tmp, totalNumChannel*sizeof(Dtype)));
}

template <typename Dtype>
void cleanupBuffer(Dtype* ptr_gpu,int count){
    hipMemset(ptr_gpu,0,count*sizeof(Dtype));
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::LoopEndCleanup_gpu(){
    int valsBuffer = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
    cleanupBuffer(this->postConv_data_gpu,valsBuffer);
    cleanupBuffer(this->postConv_grad_gpu,valsBuffer);
    cleanupBuffer(this->postBN_data_gpu,valsBuffer);
    cleanupBuffer(this->postBN_grad_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_data_gpu,valsBuffer);
    cleanupBuffer(this->postReLU_grad_gpu,valsBuffer);
}

template <typename Dtype>
__global__ void helper_computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W,int channelLimit){
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx =  (index / (H * W)) % (initChannel + growthRate * numTransition);
    if (localChannelIdx < channelLimit){
      caffe_gpu_atomic_add((xPtr[index]-batchMeanPtr[localChannelIdx]) * (xPtr[index]-batchMeanPtr[localChannelIdx]),batchVarPtr + localChannelIdx);
    }
  }
}

//variance is only used in the reverse BN process
template <typename Dtype>
void computeBatchVariance(int n,Dtype* xPtr,Dtype* batchMeanPtr,Dtype* batchVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){ 
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate; 
  helper_computeBatchVariance<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n,xPtr,batchMeanPtr,batchVarPtr,transitionIdx,numTransition,N,initChannel,growthRate,H,W,channelLimit);
  int M = N * H * W;
  caffe_gpu_scal<Dtype>(channelLimit,1.0/(M-1),batchVarPtr);
}

//ReLU: Negative_slope = 0.5
template <typename Dtype>
__global__ void ReLUForward(int n,Dtype* xPtr,Dtype* yPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index / (H * W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, fwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      yPtr[index] = xPtr[index] > 0? xPtr[index]: 0.5 * xPtr[index]; 
    }
  }
}

template <typename Dtype>
__global__ void ReLUBackward(int n,Dtype* xPtr,Dtype* dxPtr,Dtype* dyPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = initChannel + transitionIdx * growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition);
    //i.e. for transitionIdx==1, bwd both region 0 and 1
    if (localChannelIdx < channelLimit){
      dxPtr[index] = xPtr[index]>0?dyPtr[index]:0.5*dyPtr[index];
    }
  }
}

template <typename Dtype>
__global__ void ReLUReverse(int n,Dtype* yPtr,Dtype* xPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate * numTransition); 
    //i.e. for transitionIdx==1, only reverse transform region 0
    if (localChannelIdx < channelLimit){
      xPtr[index] = yPtr[index]>=0?yPtr[index]:2*yPtr[index];
    }
  }
}

template <typename Dtype>
__global__ void BNForwardInf(int n,Dtype* xPtr,Dtype* yPtr,Dtype* scalerPtr,Dtype* biasPtr,Dtype* globalMeanPtr,Dtype* globalVarPtr,int transitionIdx,int numTransition,int N,int initChannel,int growthRate,int H,int W){
  int channelLimit = transitionIdx==0?0:initChannel+(transitionIdx-1)*growthRate;
  CUDA_KERNEL_LOOP(index, n){
    int localChannelIdx = (index/(H*W)) % (initChannel + growthRate*numTransition);
    if (localChannelIdx < channelLimit){
      yPtr[index] = (scalerPtr[localChannelIdx] * ((xPtr[index]-globalMeanPtr[localChannelIdx])/sqrt(globalVarPtr[localChannelIdx] + 1e-5))) + biasPtr[localChannelIdx];
    }
  }
}

template <typename Dtype>
void composeFwdOutput(Dtype* output,Dtype* frontB,Dtype* backB,int N,int channelFront,int channelBack,int H,int W){
  for (int n=0;n<N;++n){
    int numValuesFront = channelFront*H*W;
    int numValuesBack = channelBack*H*W;
    int offsetFront = n * (channelFront + channelBack) * H * W;
    int offsetBack = offsetFront + numValuesFront;
    hipMemcpy(output+offsetFront,frontB+offsetFront,numValuesFront*sizeof(Dtype),hipMemcpyDeviceToDevice);
    hipMemcpy(output+offsetBack,backB+offsetBack,numValuesBack*sizeof(Dtype),hipMemcpyDeviceToDevice);
  }
}

template <typename Dtype>
void distributeBwdInput(Dtype* input,Dtype* frontB,Dtype* backB,int N,int channelFront,int channelBack,int H,int W){
  for (int n=0;n<N;++n){
    int numValuesFront = channelFront*H*W;
    int numValuesBack = channelBack*H*W;
    int offsetFront = n * (channelFront + channelBack) * H * W;
    int offsetBack = offsetFront + numValuesFront;
    hipMemcpy(frontB+offsetFront,input+offsetFront,numValuesFront*sizeof(Dtype),hipMemcpyDeviceToDevice);
    hipMemcpy(backB+offsetBack,input+offsetBack,numValuesBack*sizeof(Dtype),hipMemcpyDeviceToDevice);
  }
}


template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (!this->gpuInited){
      std::cout<<"Initializing GPU local"<<std::endl;
      this->GPU_Initialization();
      this->gpuInited = true;
  }
  Dtype EMA_decay = 0.99;
  clock_t begin_fwd = std::clock();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  //copy to bottom_data to buffer with stride
  int chunkSize_copy_init = this->initChannel * this->H * this->W;
  int chunkStride_copy = (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
  gpu_copy_one_to_many<Dtype>(bottom_data,this->postConv_data_gpu,this->N,chunkSize_copy_init,chunkStride_copy);
  //work in the buffer, transition by transition
  for (int transitionIdx=0;transitionIdx < this->numTransition;++transitionIdx){
      //use scaler protector before forward
      int work_n = this->N * (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;         
      //BN::type1 normal narrow channels::postConv -> postBN 
      int channelsBefore_noself = (transitionIdx==0?0:(this->initChannel + (transitionIdx - 1)*this->growthRate));
      Dtype* BN_narrow_x_ptr = this->postConv_data_gpu + channelsBefore_noself * this->H * this->W;  
      Dtype* BN_narrow_y_ptr = this->postBN_data_gpu + channelsBefore_noself * this->H * this->W;
      Dtype* BN_narrow_globalMean= this->blobs_[3*this->numTransition+transitionIdx]->mutable_gpu_data() + channelsBefore_noself;
      Dtype* BN_narrow_globalVar = this->blobs_[4*this->numTransition+transitionIdx]->mutable_gpu_data() + channelsBefore_noself;
      hipdnnTensorDescriptor_t * narrowBN_paramDesc = (transitionIdx==0?tensorDescriptor_BN_initChannel:tensorDescriptor_BN_growthRate);
      int narrow_numChannels = transitionIdx==0?this->initChannel:this->growthRate;
      Dtype* local_MeanInf = this->Mean_tmp + channelsBefore_noself;
      Dtype* local_VarInf = this->Var_tmp + channelsBefore_noself;
	      
      if (this->phase_ == TEST){
          Dtype scale_factor = this->blobs_[5*this->numTransition]->cpu_data()[0] == 0 ?
            0 : 1.0 / this->blobs_[5*this->numTransition]->cpu_data()[0];
	  caffe_gpu_scale(narrow_numChannels,scale_factor,BN_narrow_globalMean,local_MeanInf);
          caffe_gpu_scale(narrow_numChannels,scale_factor,BN_narrow_globalVar,local_VarInf);

	  /*if (transitionIdx==1){
	    std::cout<<"narrow TEST"<<std::endl;
	    print_gpuPtr(local_MeanInf,narrow_numChannels);
	    std::cout<<std::endl;
	    print_gpuPtr(local_VarInf,narrow_numChannels);
	    std::cout<<std::endl;
	  }*/

	  CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_y_ptr,
	    *narrowBN_paramDesc,
	    this->blobs_[this->numTransition+transitionIdx]->gpu_data()+channelsBefore_noself,
            this->blobs_[2*this->numTransition+transitionIdx]->gpu_data()+channelsBefore_noself,
	    local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON)
	  );
      }
      else{
          Dtype* batchMean = this->ResultSaveMean_gpu[transitionIdx] + channelsBefore_noself;
          Dtype* batchInvVar =  this->ResultSaveInvVariance_gpu[transitionIdx] + channelsBefore_noself;
	  CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_narrow_y_ptr,
	    *narrowBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data() + channelsBefore_noself,
	    this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data() + channelsBefore_noself,
	    Dtype(1),local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	  );
	  //update global Mean/Var manually
          //Mean:
	  caffe_gpu_axpby(narrow_numChannels,Dtype(1),local_MeanInf,EMA_decay,BN_narrow_globalMean);
          //Var:
	  caffe_gpu_axpby(narrow_numChannels,Dtype(1),local_VarInf,EMA_decay,BN_narrow_globalVar);

          /*if (transitionIdx==1 && (this->trainCycleIdx >=798 || (this->trainCycleIdx>=500 && this->trainCycleIdx<=502))){
	    std::cout<<"narrow TRAIN"<<std::endl;
	    print_gpuPtr(local_VarInf,narrow_numChannels);
	    std::cout<<std::endl;
	    print_gpuPtr(local_VarInf,narrow_numChannels);
	    std::cout<<std::endl;
	  }*/

      }
      //BN :: type2: wide channels, for anything prior
      if (transitionIdx > 0){
        hipdnnTensorDescriptor_t* wideBN_paramDesc = this->tensorDescriptor_BN_wide[transitionIdx]; 
	Dtype* BN_wide_x_ptr = this->postReLU_data_gpu;
	Dtype* BN_wide_y_ptr = this->postBN_data_gpu;
	Dtype* BN_wide_globalMean = this->blobs_[3*this->numTransition+transitionIdx]->mutable_gpu_data();
	Dtype* BN_wide_globalVar = this->blobs_[4*this->numTransition+transitionIdx]->mutable_gpu_data();
        Dtype* local_MeanInf = this->Mean_tmp;
	Dtype* local_VarInf = this->Var_tmp;
	int wide_numChannels = transitionIdx==0?0:this->initChannel+this->growthRate*(transitionIdx-1);   
        if (this->phase_ == TEST){
	  Dtype scale_factor = this->blobs_[5*this->numTransition]->cpu_data()[0] == 0 ?
            0 : 1 / this->blobs_[5*this->numTransition]->cpu_data()[0];
	  caffe_gpu_scale(wide_numChannels,scale_factor,BN_wide_globalMean,local_MeanInf);
          caffe_gpu_scale(wide_numChannels,scale_factor,BN_wide_globalVar,local_VarInf);
	  
	  /*if (transitionIdx==1){
	    std::cout<<"wide TEST"<<std::endl;
	    print_gpuPtr(local_MeanInf,wide_numChannels);
	    std::cout<<std::endl;
	    print_gpuPtr(local_VarInf,wide_numChannels);
	    std::cout<<std::endl;
	  }*/

	  CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_y_ptr,
	    *wideBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
            this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON)
	  );
	}
	else {
	  Dtype* batchMean = this->ResultSaveMean_gpu[transitionIdx];
	  Dtype* batchInvVar = this->ResultSaveInvVariance_gpu[transitionIdx];
          CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_x_ptr,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BN_wide_y_ptr,
	    *wideBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	    this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    Dtype(1),local_MeanInf,local_VarInf,HIPDNN_BN_MIN_EPSILON,
	    batchMean,batchInvVar)
	  );
	  //update global Mean/Var manually
          //Mean:
	  caffe_gpu_axpby(wide_numChannels,Dtype(1),local_MeanInf,EMA_decay,BN_wide_globalMean);
          //Var:
	  caffe_gpu_axpby(wide_numChannels,Dtype(1),local_VarInf,EMA_decay,BN_wide_globalVar);
          /*if (transitionIdx==1 && ((this->trainCycleIdx >= 798) || (this->trainCycleIdx<=502 && this->trainCycleIdx>=500))){
	    std::cout<<"wide TRAIN"<<std::endl;
	    print_gpuPtr(batchMean,wide_numChannels);
	    std::cout<<std::endl;
	    print_gpuPtr(batchInvVar,wide_numChannels);
	    std::cout<<std::endl;
	  }*/
        }
      }
      //cache postReLU to cache region
      int cache_size = this->N * (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
      CUDA_CHECK(hipMemcpy(this->postReLU_cache_cpu[transitionIdx],this->postReLU_data_gpu,cache_size*sizeof(Dtype),hipMemcpyDeviceToHost));
      //ReLU
      Dtype* ReLU_x_ptr = this->postBN_data_gpu;
      Dtype* ReLU_y_ptr = this->postReLU_data_gpu;
      ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(work_n), CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_ptr,ReLU_y_ptr,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
      //Convolution
      int delayChannel = this->initChannel + this->growthRate * transitionIdx;
      Dtype* conv_x_local = this->postReLU_data_gpu;
      Dtype* conv_y_local = this->postConv_data_gpu + delayChannel * this->H * this->W;
      CUDNN_CHECK(hipdnnConvolutionForward(*(this->cudnnHandlePtr),
	cudnn::dataType<Dtype>::one,
	*(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	*(this->filterDescriptorVec[transitionIdx]),
	this->blobs_[transitionIdx]->gpu_data(),
	*(this->conv_Descriptor),HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
	this->workspace,this->workspace_size_bytes,cudnn::dataType<Dtype>::zero,
	*(this->tensorDescriptor_conv_y),conv_y_local	
	)		      
      );
      //this->logInternal_gpu("TClog",transitionIdx,true,false);
  } 
  if (this->phase_ == TRAIN){
    this->blobs_[5*this->numTransition]->mutable_cpu_data()[0] *= EMA_decay;
    this->blobs_[5*this->numTransition]->mutable_cpu_data()[0] += 1;
    this->trainCycleIdx+=1;
  }
  //deploy top data
  composeFwdOutput(top[0]->mutable_gpu_data(),this->postReLU_data_gpu,this->postConv_data_gpu,this->N,this->initChannel+this->growthRate*(this->numTransition-1),this->growthRate,this->H,this->W);
  //clock_t end_fwd = std::clock();
  //double elapsed_fwd = double(end_fwd - begin_fwd) / CLOCKS_PER_SEC;
  //std::cout<<"elapsed fwd gpu:"<<elapsed_fwd<<std::endl;
  //this->logInternal_gpu("TClog",-1,false,false);
  //this->logInternal_gpu("TClog");
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    if (!this->gpuInited){
	this->GPU_Initialization();
    	this->gpuInited = true;
    }
    //clock_t begin_bwd = std::clock();
    //assuming buffers store already computed value, always propagate down
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    //deploy top diff to buffer
    distributeBwdInput(top[0]->mutable_gpu_diff(),this->postReLU_grad_gpu,this->postConv_grad_gpu,this->N,this->initChannel+this->growthRate*(this->numTransition-1),this->growthRate,this->H,this->W);
    //Backward, transition by transition
    for (int transitionIdx=this->numTransition-1;transitionIdx>=0;--transitionIdx){
        int channelsBefore_self = this->initChannel + transitionIdx * this->growthRate;
        int channelsBefore_noself = transitionIdx>0?(this->initChannel + (transitionIdx - 1) * this->growthRate):0;
	//Conv
        Dtype* filterGrad_local = this->blobs_[transitionIdx]->mutable_gpu_diff();
	const Dtype* filterData_local =this->blobs_[transitionIdx]->gpu_data();
	Dtype* conv_x_local = postReLU_data_gpu;
	Dtype* conv_dy_local = postConv_grad_gpu + channelsBefore_self * this->H * this->W;
	//Conv w.r.t. filter
	CUDNN_CHECK(hipdnnConvolutionBackwardFilter(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one, 
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::zero,
	  *(this->filterDescriptorVec[transitionIdx]),filterGrad_local	  
	  )		
	);
	//Conv w.r.t. x
	CUDNN_CHECK(hipdnnConvolutionBackwardData(*(this->cudnnHandlePtr),
	  cudnn::dataType<Dtype>::one,
	  *(this->filterDescriptorVec[transitionIdx]),filterData_local,
	  *(this->tensorDescriptor_conv_y),conv_dy_local,
	  *(this->conv_Descriptor),HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
	  this->workspace,this->workspace_size_bytes,
	  cudnn::dataType<Dtype>::one,
	  *(this->tensorDescriptorVec_conv_x[transitionIdx]),postReLU_grad_gpu
	  )		
	);
	//ReLU Bwd, for any j <= transitionIdx
	int work_n = this->N * (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;
	Dtype* ReLU_x_local = postBN_data_gpu;
	Dtype* ReLU_dy_local = postReLU_grad_gpu;
        Dtype* ReLU_dx_local = postBN_grad_gpu;	
	ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,ReLU_x_local,ReLU_dx_local,ReLU_dy_local,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
        //use cache to restore postReLU region data
	int cache_size = this->N * (this->initChannel+this->growthRate*this->numTransition) * this->H * this->W; 
        CUDA_CHECK(hipMemcpy(postReLU_data_gpu,postReLU_cache_cpu[transitionIdx],cache_size*sizeof(Dtype),hipMemcpyHostToDevice)); 
	//BN Bwd, type2, wide
	if (transitionIdx > 0){
	  Dtype* BNwide_x_local = this->postReLU_data_gpu;
	  Dtype* BNwide_dx_local = this->postReLU_grad_gpu;
	  Dtype* BNwide_dy_local = this->postBN_grad_gpu;
	  Dtype* saveMeanwide_local = this->ResultSaveMean_gpu[transitionIdx]; 
	  Dtype* saveInvVarwide_local = this->ResultSaveInvVariance_gpu[transitionIdx];
	  hipdnnTensorDescriptor_t * BNwideparam_desc = this->tensorDescriptor_BN_wide[transitionIdx];
	
	  CUDNN_CHECK(hipdnnBatchNormalizationBackward(*(this->cudnnHandlePtr),
	    HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_x_local,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_dy_local,
	    *(this->tensorDescriptorVec_conv_x[transitionIdx-1]),BNwide_dx_local,
	    *BNwideparam_desc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	    this->blobs_[this->numTransition + transitionIdx]->mutable_gpu_diff(),
	    this->blobs_[2*this->numTransition + transitionIdx]->mutable_gpu_diff(),
	    HIPDNN_BN_MIN_EPSILON,saveMeanwide_local,saveInvVarwide_local
	    )		
	  );
	}
	//BN Bwd, type1, narrow
        Dtype* BNnarrow_x_local = this->postConv_data_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* BNnarrow_dx_local = this->postConv_grad_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* BNnarrow_dy_local = this->postBN_grad_gpu + channelsBefore_noself * this->H * this->W;
	Dtype* saveMeannarrow_local = this->ResultSaveMean_gpu[transitionIdx] + channelsBefore_noself;
	Dtype* saveInvVarnarrow_local = this->ResultSaveInvVariance_gpu[transitionIdx] + channelsBefore_noself;
        hipdnnTensorDescriptor_t * BNnarrowparam_desc = (transitionIdx==0)?tensorDescriptor_BN_initChannel : tensorDescriptor_BN_growthRate;
        CUDNN_CHECK(hipdnnBatchNormalizationBackward(*(this->cudnnHandlePtr),
	  HIPDNN_BATCHNORM_SPATIAL,
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	  cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_x_local,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_dy_local,
	  *(this->tensorDescriptorVec_narrow[transitionIdx]),BNnarrow_dx_local,
	  *BNnarrowparam_desc,
	  this->blobs_[this->numTransition + transitionIdx]->gpu_data() + channelsBefore_noself,
	  this->blobs_[this->numTransition + transitionIdx]->mutable_gpu_diff() + channelsBefore_noself,
	  this->blobs_[2*this->numTransition + transitionIdx]->mutable_gpu_diff() + channelsBefore_noself,
	  HIPDNN_BN_MIN_EPSILON,saveMeannarrow_local,saveInvVarnarrow_local
	  )		
	);	
	//BN data region reverse using ReLUReverse
        Dtype* BNregion_reverse_y_local = this->postReLU_data_gpu;
	Dtype* BNregion_reverse_x_local = this->postBN_data_gpu;
	ReLUReverse<Dtype><<<CAFFE_GET_BLOCKS(work_n),CAFFE_CUDA_NUM_THREADS>>>(work_n,BNregion_reverse_y_local,BNregion_reverse_x_local,transitionIdx,this->numTransition,this->N,this->initChannel,this->growthRate,this->H,this->W);
	
	//this->logInternal_gpu("TClog",transitionIdx,true,false);
        //this->logInternal_gpu("TClog",transitionIdx,true,true);
    }
    //deploy buffer to bottom diff
    //this->logInternal_gpu("TClog",-1,false,false);
    int chunkSize_copy_init = this->initChannel * this->H * this->W;
    int chunkStride_copy = (this->initChannel + this->numTransition * this->growthRate) * this->H * this->W;
    gpu_copy_many_to_one(postConv_grad_gpu,bottom_diff,this->N,chunkSize_copy_init,chunkStride_copy);
    this->LoopEndCleanup_gpu();
    //clock_t end_bwd = std::clock();
    //double elapsed_bwd = double(end_bwd - begin_bwd) / CLOCKS_PER_SEC;
    //std::cout<<"elapsed bwd time:"<<elapsed_bwd<<std::endl;
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu_public(const vector<Blob<Dtype>*>& bottom,const vector<Blob<Dtype>*>& top){
  this->Forward_gpu(bottom,top);
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu_public(const vector<Blob<Dtype>*>& top,const vector<bool>& propagate_down,const vector<Blob<Dtype>*>& bottom){
  this->Backward_gpu(top,propagate_down,bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

}  // namespace caffe
