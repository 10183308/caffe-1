#include <vector>

#include "caffe/layers/DenseBlock_layer.hpp"

namespace caffe {

template <typename Dtype>
void gpu_copy_one_to_many(const Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_input,int chunkStride_output){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
	const Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkSize_input; 
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkStride_output;
        CUDA_CHECK(hipMemcpy(outPtr_local,inPtr_local,chunkSize_input * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void gpu_copy_many_to_one(const Dtype* inPtr_gpu,Dtype* outPtr_gpu,int numChunks,int chunkSize_output,int chunkStride_input){
    for (int chunkIdx=0;chunkIdx<numChunks;++chunkIdx){
        const Dtype* inPtr_local = inPtr_gpu + chunkIdx*chunkStride_input;
	Dtype* outPtr_local = outPtr_gpu + chunkIdx*chunkSize_output;
	CUDA_CHECK(hipMemcpy(inPtr_local,outPtr_local,chunkSize_output * sizeof(Dtype),hipMemcpyDeviceToDevice));
    }
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  //copy to bottom_data to buffer with stride
  int chunkSize_copy = this->initChannel * this->H * this->W;
  int chunkStride_copy = (this->initChannel + this->growthRate * this->numTransition) * this->H * this->W;
  gpu_copy_one_to_many<Dtype>(bottom_data,this->postConv_data_gpu,this->N,chunkSize_copy,chunkStride_copy);
  //work in the buffer, transition by transition
  for (int transitionIdx=0;transitionIdx < this->numTransition;++transitionIdx){
      //BN and ReLU
      int channelsBefore_noself = (transitionIdx==0?0:(this->initChannel + (transitionIdx - 1)*this->growthRate));
      Dtype* BN_x_ptr = this->postConv_data_gpu + channelsBefore_noself * this->H * this->W;  
      Dtype* BN_y_ptr = this->postBN_data_gpu + channelsBefore_noself * this->H * this->W;
      Dtype* ReLU_y_ptr = this->postReLU_data_gpu + channelsBefore_noself * this->H * this->W;
      //BN
      Dtype* BN_mean_local = this->ResultRunningMean_gpu + channelsBefore_noself;
      Dtype* BN_var_local = this->ResultRunningVariance_gpu + channelsBefore_noself;
      hipdnnTensorDescriptor_t * localBN_paramDesc = (transitionIdx==0?tensorDescriptor_BN_initChannel:tensorDescriptor_BN_growthRate);
      if (this->phase_ == TEST){
          CUDNN_CHECK(hipdnnBatchNormalizationForwardInference(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_y_ptr,
	    *localBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
            this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    BN_mean_local,BN_var_local,HIPDNN_BN_MIN_EPSILON)
	  );
      }
      else{
          Dtype* resultSaveMean_local = this->ResultSaveMean_gpu + channelsBefore_noself;
          Dtype* resultSaveInvVariance_local =  this->ResultSaveInvVariance_gpu + channelsBefore_noself;
	  double EMA_factor = 1.0/(1+this->trainCycleIdx);	  
	  CUDNN_CHECK(hipdnnBatchNormalizationForwardTraining(
	    *(this->cudnnHandlePtr),HIPDNN_BATCHNORM_SPATIAL,
	    cudnn::dataType<Dtype>::one,cudnn::dataType<Dtype>::zero,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_x_ptr,
	    *(this->tensorDescriptorVec_narrow[transitionIdx]),BN_y_ptr,
	    *localBN_paramDesc,
	    this->blobs_[this->numTransition + transitionIdx]->gpu_data(),
	    this->blobs_[2 * this->numTransition + transitionIdx]->gpu_data(),
	    EMA_factor,BN_mean_local,BN_var_local,HIPDNN_BN_MIN_EPSILON,
	    resultSaveMean_local,resultSaveInvVariance_local)
	  );
	  this->trainCycleIdx += 1;
      } 
      //ReLU
      CUDNN_CHECK(hipdnnActivationForward(*(this->cudnnHandlePtr),
	*(this->activationDesc), cudnn::dataType<Dtype>::one, 
	*(this->tensorDescriptorVec_narrow[transitionIdx]),BN_y_ptr,
	cudnn::dataType<Dtype>::zero,
	*(this->tensorDescriptorVec_narrow[transitionIdx]),ReLU_y_ptr)
      );
      //Convolution
      int delayChannel = this->initChannel + this->growthRate * transitionIdx;
      Dtype* conv_x_local = postReLU_data_gpu;
      Dtype* conv_y_local = postConv_data_gpu + delayChannel * this->H * this->W;
      CUDNN_CHECK(hipdnnConvolutionForward(*(this->cudnnHandlePtr),
	cudnn::dataType<Dtype>::one,
	*(this->tensorDescriptorVec_conv_x[transitionIdx]),conv_x_local,
	*(this->filterDescriptorVec[transitionIdx]),
	this->blobs_[transitionIdx]->gpu_data(),
	*(this->conv_Descriptor),HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
	this->workspace,this->workspace_size_bytes,cudnn::dataType<Dtype>::zero,
	*(this->tensorDescriptor_conv_y),conv_y_local	
	)		      
      ); 
  } 
  //change top data
  int resultChannelGap = this->initChannel + this->growthRate * (this->numTransition - 1);
  Dtype* resultBuffer_ptr = postConv_data_gpu + resultChannelGap * this->H * this->W;
  gpu_copy_many_to_one<Dtype>(resultBuffer_ptr,top_data,this->N,chunkSize_copy,chunkStride_copy);
}

template <typename Dtype>
void DenseBlockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
  } 
}

INSTANTIATE_LAYER_GPU_FUNCS(DenseBlockLayer);

}  // namespace caffe
